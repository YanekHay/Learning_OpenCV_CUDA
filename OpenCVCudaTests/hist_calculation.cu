#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hist_calculation_CUDA.h"

__global__ void calc_histogram_CUDA(unsigned char* image, int* histogram);
__global__ void calc_histogram_CUDA(unsigned char* image, int channels, int* B_hist, int* G_hist, int* R_hist);

void hist_calculation_CUDA(unsigned char* image, int height, int width, int channels, int* hist_grayscale) {
	unsigned char* cuda_image = NULL;
	int* cuda_hist = NULL;

	int image_size = height * width * channels;
	int hist_size = sizeof(int) * BIN_COUNT;
	//allocate
	hipMalloc((void**)&cuda_image, image_size);
	hipMalloc((void**)&cuda_hist, hist_size);

	//copy
	hipMemcpy(cuda_image, image, image_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_hist, hist_grayscale, hist_size, hipMemcpyHostToDevice);

	//
	dim3 grid_image(width, height);
	calc_histogram_CUDA<<<grid_image, 1>>> (cuda_image, cuda_hist);
	hipMemcpy(hist_grayscale, cuda_hist, hist_size, hipMemcpyDeviceToHost);

	//Free up GPU
	hipFree(cuda_image);
	hipFree(cuda_hist);

}



void hist_calculation_CUDA(unsigned char* image, int height, int width, int channels, int* B_hist, int* G_hist, int* R_hist) {
	unsigned char* cuda_image = NULL;
	int* cuda_B_hist = NULL;
	int* cuda_G_hist = NULL;
	int* cuda_R_hist = NULL;

	int image_size = height * width * channels;
	int hist_size = sizeof(int) * BIN_COUNT;

	//allocate
	hipMalloc((void**)&cuda_image, image_size);
	hipMalloc((void**)&cuda_B_hist, hist_size);
	hipMalloc((void**)&cuda_G_hist, hist_size);
	hipMalloc((void**)&cuda_R_hist, hist_size);


	hipMemcpy(cuda_image, image, image_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_B_hist, B_hist, hist_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_G_hist, G_hist, hist_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_R_hist, R_hist, hist_size, hipMemcpyHostToDevice);


	dim3 grid_image(width, height);
	calc_histogram_CUDA << <grid_image, 1 >> > (cuda_image, channels, cuda_B_hist, cuda_G_hist, cuda_R_hist);
	hipMemcpy(B_hist, cuda_B_hist, hist_size, hipMemcpyDeviceToHost);
	hipMemcpy(G_hist, cuda_G_hist, hist_size, hipMemcpyDeviceToHost);
	hipMemcpy(R_hist, cuda_R_hist, hist_size, hipMemcpyDeviceToHost);

	//Free up GPU
	hipFree(cuda_image);
	hipFree(cuda_B_hist);
	hipFree(cuda_G_hist);
	hipFree(cuda_R_hist);


}

__global__ void calc_histogram_CUDA(unsigned char* image, int* histogram) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = x + y * gridDim.x;

	atomicAdd(&histogram[image[idx]], 1);
}


__global__ void calc_histogram_CUDA(unsigned char* image, int channels, int* B_hist, int* G_hist, int* R_hist) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * channels;

	atomicAdd(&B_hist[image[idx]], 1);
	atomicAdd(&G_hist[image[idx+1]], 1);
	atomicAdd(&R_hist[image[idx+2]], 1);

}